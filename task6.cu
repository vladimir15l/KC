/*implementation of a small neural network*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <hipblas.h>


hipblasHandle_t handle;
// Just a sigmoid
__global__ void sigmoid(float* x) {
    int idx = threadIdx.x;
    x[idx] = exp(x[idx]) / (1 + exp(x[idx]));
}

// This class implements a fully connected layer
class Linear {
    float* weight;
    float* bias;
    int in_features;
    int out_features;
public:
    Linear() {
        weight = NULL;
        bias = NULL;
        in_features = 0;
        out_features = 0;
    };
    Linear(int in, int out) {
        weight = NULL;
        bias = NULL;
        in_features = in;
        out_features = out;
    }
    //initializes weights and bias
    void initializer(FILE* weights){
        float* w = (float*)malloc(in_features * out_features * sizeof(float));
        float* b = (float*)malloc(out_features * sizeof(float));
        fread(w, sizeof(float), in_features*out_features, weights);
        fread(b, sizeof(float), out_features, weights);
        hipMalloc((void**)&weight, in_features * out_features * sizeof(float));
        hipMalloc((void**)&bias, out_features * sizeof(float));
        hipMemcpy(weight, w, in_features * out_features * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(bias, b, out_features * sizeof(float), hipMemcpyHostToDevice);
        free(w);
        free(b);
    }
    // the vector with the input data is multiplied with the weight matrix
    float* operator() (float* x) {
        const float a = 1;
        hipblasSgemv(handle, HIPBLAS_OP_T, in_features, out_features, &a, weight, in_features, x, 1, &a, bias, 1);
        hipblasScopy(handle, out_features, bias, 1, x, 1);  
        return x;
    }
    ~Linear() {
        if (weight)
            hipFree(weight);
        if (bias)
            hipFree(bias);
    }
};

// A neural network model with three fully connected layers
class Net {
    Linear fc1;
    Linear fc2;
    Linear fc3;
    // direct dissemination of information
    float forward(float* x) {
        sigmoid<<<1, 256>>>(fc1(x));
        sigmoid<<<1, 16>>>(fc2(x));
        sigmoid<<<1, 1>>>(fc3(x));
        float result;
        hipMemcpy(&result, x, sizeof(float), hipMemcpyDeviceToHost);
        return result;
    }
public:
    Net(int in, int middle1, int middle2) {
        hipblasCreate(&handle);
        FILE* weight = fopen("weight.npy", "rb");
        if (weight == NULL) {
            printf(" Error writing in weight file\n");
            exit(1);
        }
        fc1 = Linear(in, middle1);
        fc2 = Linear(middle1, middle2);
        fc3 = Linear(middle2, 1);
        fc1.initializer(weight);
        fc2.initializer(weight);
        fc3.initializer(weight);
    }
    // Launching a neural network. Reading input data from a file
    // and starting a direct flow of information
    float operator() (char* file, int size) {        
        FILE* input = fopen(file, "rb");
        if (input == NULL) {
            printf(" Error writing in input file\n");
            exit(1);
        }
        float* input_layer = (float*)malloc(size * sizeof(float));  
    
        if(input_layer) fread(input_layer, sizeof(float), size, input);

        float* d_layer;
        hipMalloc((void**)&d_layer, size*sizeof(float));
        hipMemcpy(d_layer, input_layer, size*sizeof(float), hipMemcpyHostToDevice);
	    free(input_layer);
        return forward(d_layer);
    }
    ~Net(){
        hipblasDestroy(handle);
    }
};

int main() {
    int size = 1024;
    Net net = Net(1024, 256, 16);    
    float result = net("input.npy", size);
    printf("%lf\n\n", result);    
    return 0;
}